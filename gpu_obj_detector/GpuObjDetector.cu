#include "hip/hip_runtime.h"
/*
 * GpuObjDetector.cpp
 *
 *  Created on: May 25, 2013
 *      Author: olehp
 */

#include "GpuObjDetector.h"
#include "gpu_utils.h"
#include "utils.h"

#include <thrust/scan.h>

#include <iostream>
#include <algorithm>

#define MAX_THREAD 416

using namespace std;
using namespace thrust;



__constant__ __align__(4) char stage_buf[sizeof(Stage)];


inline int GetNumBlocks(int n) {
	return ceilf((float) n / MAX_THREAD);
}

__device__ inline int RectSum(const int* ii, int x, int y, int w, int h, int ii_width) {

	return ii[y * ii_width + x] +
		   ii[(y + h) * ii_width + x + w] -
		   ii[y * ii_width + x + w] -
		   ii[(y + h) * ii_width + x];
}



__global__ void kernel_ii_rows(const int *matr, int *result, int *sq_result, int rows, int cols) {

	int row = threadIdx.x + blockIdx.x * blockDim.x;

	int img_start_offset = row * cols;
	int ii_start_offset = (row + 1) * (cols + 1) + 1;

	int val;
	int i;
	int cur_sum = 0, cur_sq_sum = 0;

	if (row < rows) {

		for (i = 0; i < cols; i++) {
			val = matr[img_start_offset + i];
			cur_sum += val;
			cur_sq_sum += (val * val);

			result[ii_start_offset + i] = cur_sum;
			sq_result[ii_start_offset + i] = cur_sq_sum;
		}
	}
}

__global__ void kernel_ii_cols(int *result, int *sq_result, int rows, int cols) {

	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int i;
	int ii_start_offset = (cols + 1) + col + 1;
	if (col < cols) {
		for (i = 1; i < rows; i++) {
			result[ii_start_offset + i * (cols + 1)] += result[ii_start_offset + (i - 1) * (cols + 1)];
			sq_result[ii_start_offset + i * (cols + 1)] += sq_result[ii_start_offset + (i - 1) * (cols + 1)];

		}
	}
}


void GpuObjDetector::GpuComputeII() {

	HANDLE_ERROR(hipMemset(dev_ii, 0, ii_mem_size));
	HANDLE_ERROR(hipMemset(dev_ii2, 0, ii_mem_size));

	dim3 block(512);
	dim3 grid_rows(ceil(img_height / 512.0));
	dim3 grid_cols(ceil(img_width / 512.0));

	kernel_ii_rows<<<grid_rows, block>>>(dev_img, dev_ii, dev_ii2, img_height, img_width);
	kernel_ii_cols<<<grid_cols, block>>>(dev_ii, dev_ii2, img_height, img_width);
}

__global__ void kernel_precalc_inv_and_stddev(const ScaledRectangle *subwindows,
											  const int *ii,
											  const int *ii2,
											  float *invs,
											  float *std_devs,
											  int ii_width,
											  int num) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	if (i < num) {

		int x = subwindows[i].x;
		int y = subwindows[i].y;
		int width = subwindows[i].w;
		int height = subwindows[i].h;

		float inv = 1.f / (height * width);
		float mean = RectSum(ii, x, y, width, height, ii_width) * inv;
		float variance = abs(RectSum(ii2, x, y, width, height, ii_width) * inv - mean * mean);

		float std_dev = sqrtf(variance);

		invs[i] = inv;
		std_devs[i] = std_dev;
	}
}

void GpuObjDetector::PrecalcInvAndStdDev() {
	kernel_precalc_inv_and_stddev<<<GetNumBlocks(all_subwindows.size()),
									MAX_THREAD>>>(dev_subwindows_in,
												  dev_ii,
												  dev_ii2,
												  dev_inv_in,
												  dev_std_dev_in,
												  img_width + 1,
												  num);
}

__global__ void kernel_compact_arrays(const ScaledRectangle *subwindows_in,
									  ScaledRectangle *subwindows_out,
									  const float *invs_in,
									  float *invs_out,
									  const float *std_dev_in,
									  float *std_dev_out,
									  const int *is_valid,
									  const int *indexes,
									  int num) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	if ((i < num) && is_valid[i]) {

		int pos = indexes[i] - 1;

		subwindows_out[pos] = subwindows_in[i];
		invs_out[pos] = invs_in[i];
		std_dev_out[pos] = std_dev_in[i];

	}
}


void GpuObjDetector::CompactArrays(int& num_subwindows) {
//	CUDPPHandle scan_plan;
//	cudppPlan(lib, &scan_plan, scan_conf, num_subwindows, 1, 0);
//	cudppScan(scan_plan, dev_indexes, dev_is_valid, num_subwindows);

	device_ptr<int> t_dev_is_valid = device_pointer_cast(dev_is_valid);
	device_ptr<int> t_dev_indexes = device_pointer_cast(dev_indexes);


	inclusive_scan(t_dev_is_valid, t_dev_is_valid + num_subwindows, t_dev_indexes);


	kernel_compact_arrays<<<GetNumBlocks(num_subwindows), MAX_THREAD>>>(dev_subwindows_in,
																		dev_subwindows_out,
																		dev_inv_in,
																		dev_inv_out,
																		dev_std_dev_in,
																		dev_std_dev_out,
																		dev_is_valid,
																		dev_indexes,
																		num_subwindows);

	hipMemcpy(&num_subwindows, (dev_indexes + num_subwindows - 1), sizeof(int), hipMemcpyDeviceToHost);

	swap(dev_subwindows_in, dev_subwindows_out);
	swap(dev_inv_in, dev_inv_out);
	swap(dev_std_dev_in, dev_std_dev_out);

//	cudppDestroyPlan(scan_plan);
}

__global__ void kernel_detect_objs(const int *ii,
								   const int *ii2,
								   const float *invs,
								   const float *std_devs,
								   int ii_width,
								   int ii_height,
								   const ScaledRectangle *subwindows,
								   int *is_valid,
								   int num_subwindows) {
	// 244 216 123 123 6.19174

	int i_subwindow = threadIdx.x + blockIdx.x * blockDim.x;

	if (!(i_subwindow < num_subwindows)) return;

	float scale = subwindows[i_subwindow].scale;
	int x = subwindows[i_subwindow].x;
	int y = subwindows[i_subwindow].y;

	float inv = invs[i_subwindow];

	float std_dev = std_devs[i_subwindow];

	Stage &stage = (Stage&)stage_buf;

	float tree_sum = 0;

	for (int j = 0; j < HAAR_MAX_TREES; j++) {
		Tree& tree = stage.trees[j];
		if (!tree.valid) break;

		float rects_sum = 0;

		for (int k = 0; k < HAAR_MAX_RECTS; k++) {
			WeightedRectangle &rect = tree.feature.rects[k];
			if (rect.wg == 0) break;

			rects_sum = rects_sum + RectSum(ii, x + (int)(rect.x * scale),
												y + (int)(rect.y * scale),
												(int)(rect.w * scale),
												(int)(rect.h * scale), ii_width) * rect.wg;
		}

		tree_sum += ((rects_sum * inv < tree.threshold * std_dev) ? tree.left_val : tree.right_val);
	}


	is_valid[i_subwindow] = (tree_sum >= stage.threshold);
}

void GpuObjDetector::DetectAtSubwindows(vector<Rectangle>& objs) {

	int num_subwindows = all_subwindows.size();
	HANDLE_ERROR(hipMemcpy(dev_subwindows_in, &all_subwindows[0], sizeof(ScaledRectangle) * num_subwindows, hipMemcpyHostToDevice));

	PrecalcInvAndStdDev();

	for (int i = 0; i < HAAR_MAX_STAGES; i++) {

		if (num_subwindows == 0) break;

		HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(stage_buf), &haar_cascade.stages[i], sizeof(Stage)));

		kernel_detect_objs<<<GetNumBlocks(num_subwindows), MAX_THREAD>>>(
											dev_ii,
											dev_ii2,
											dev_inv_in,
											dev_std_dev_in,
											img_width + 1,
											img_height + 1,
											dev_subwindows_in,
											dev_is_valid,
											num_subwindows);

		CompactArrays(num_subwindows);
	}

	vector<ScaledRectangle> result(num_subwindows);
	HANDLE_ERROR(hipMemcpy(&result[0], dev_subwindows_in, sizeof(ScaledRectangle) * num_subwindows, hipMemcpyDeviceToHost));

	objs.assign(result.begin(), result.end());

}

GpuObjDetector::GpuObjDetector(int w, int h, HaarCascade& cascade) :
	img_width(w),
	img_height(h),
	haar_cascade(cascade) {

	img_mem_size = img_width * img_height * sizeof(int);
	ii_mem_size = (img_width + 1) * (img_height + 1) * sizeof(int);

	HANDLE_ERROR(hipMalloc(&dev_img, img_mem_size));
	HANDLE_ERROR(hipMalloc(&dev_ii, ii_mem_size));
	HANDLE_ERROR(hipMalloc(&dev_ii2, ii_mem_size));

	PrecalcSubwindows();

	HANDLE_ERROR(hipMalloc(&dev_subwindows_in, sizeof(ScaledRectangle) * all_subwindows.size()));
	HANDLE_ERROR(hipMalloc(&dev_subwindows_out, sizeof(ScaledRectangle) * all_subwindows.size()));
	HANDLE_ERROR(hipMalloc(&dev_is_valid, sizeof(int) * all_subwindows.size()));
	HANDLE_ERROR(hipMalloc(&dev_indexes, sizeof(int) * all_subwindows.size()));

	HANDLE_ERROR(hipMalloc(&dev_inv_in, sizeof(float) * all_subwindows.size()));
	HANDLE_ERROR(hipMalloc(&dev_inv_out, sizeof(float) * all_subwindows.size()));
	HANDLE_ERROR(hipMalloc(&dev_std_dev_in, sizeof(float) * all_subwindows.size()));
	HANDLE_ERROR(hipMalloc(&dev_std_dev_out, sizeof(float) * all_subwindows.size()));


//	cudppCreate(&lib);
//	scan_conf.op = CUDPP_ADD;
//	scan_conf.datatype = CUDPP_INT;
//	scan_conf.algorithm = CUDPP_SCAN;
//	scan_conf.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_INCLUSIVE;

}


void GpuObjDetector::Detect(const int *g_img, std::vector<Rectangle>& objs) {

	HANDLE_ERROR(hipMemcpy(dev_img, g_img, img_mem_size, hipMemcpyHostToDevice));
	GpuComputeII();
	DetectAtSubwindows(objs);
}

void GpuObjDetector::PrecalcSubwindows() {

	float scale = 1.0;

	int start_width = haar_cascade.window_width;
	int start_height = haar_cascade.window_height;

	int width = start_width;
	int height = start_height;

	while (OR_MIN(width, height) <= OR_MIN(img_width, img_height)) {

		int x_step = 5;//OR_MAX(1, OR_MIN(4, width / 10));
		int y_step = 5;//OR_MAX(1, OR_MIN(4, height / 10));

		for (int y = 0; y < img_height - height; y += y_step) {
			for (int x = 0; x < img_width - width; x += x_step) {
				all_subwindows.push_back(ScaledRectangle(x, y, width, height, scale));
			}
		}

		scale = scale * 1.2;
		width = (int)(start_width * scale);
		height = (int)(start_height * scale);
	}
}

GpuObjDetector::~GpuObjDetector() {
	HANDLE_ERROR(hipFree(dev_img));
	HANDLE_ERROR(hipFree(dev_ii));
	HANDLE_ERROR(hipFree(dev_ii2));
	HANDLE_ERROR(hipFree(dev_subwindows_in));
	HANDLE_ERROR(hipFree(dev_subwindows_out));
	HANDLE_ERROR(hipFree(dev_is_valid));
	HANDLE_ERROR(hipFree(dev_indexes));

	HANDLE_ERROR(hipFree(dev_inv_in));
	HANDLE_ERROR(hipFree(dev_inv_out));
	HANDLE_ERROR(hipFree(dev_std_dev_in));
	HANDLE_ERROR(hipFree(dev_std_dev_out));

//	cudppDestroy(lib);

}
