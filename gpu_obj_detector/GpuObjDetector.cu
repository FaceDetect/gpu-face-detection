#include "hip/hip_runtime.h"
/*
 * GpuObjDetector.cpp
 *
 *  Created on: May 25, 2013
 *      Author: olehp
 */

#include "GpuObjDetector.h"
#include "gpu_utils.h"
#include "utils.h"

#include <iostream>
#include <algorithm>

#define MAX_THREAD 416

using namespace std;


__constant__ __align__(4) char stage_buf[sizeof(Stage)];

__device__ inline int RectSum(const int* ii, int x, int y, int w, int h, int ii_width) {

	return ii[y * ii_width + x] +
		   ii[(y + h) * ii_width + x + w] -
		   ii[y * ii_width + x + w] -
		   ii[(y + h) * ii_width + x];
}



__global__ void kernel_ii_rows(const int *matr, int *result, int *sq_result, int rows, int cols) {

	int row = threadIdx.x + blockIdx.x * blockDim.x;

	int img_start_offset = row * cols;
	int ii_start_offset = (row + 1) * (cols + 1) + 1;

	int val;
	int i;
	int cur_sum = 0, cur_sq_sum = 0;

	if (row < rows) {

		for (i = 0; i < cols; i++) {
			val = matr[img_start_offset + i];
			cur_sum += val;
			cur_sq_sum += (val * val);

			result[ii_start_offset + i] = cur_sum;
			sq_result[ii_start_offset + i] = cur_sq_sum;
		}
	}
}

__global__ void kernel_ii_cols(int *result, int *sq_result, int rows, int cols) {

	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int i;
	int ii_start_offset = (cols + 1) + col + 1;
	if (col < cols) {
		for (i = 1; i < rows; i++) {
			result[ii_start_offset + i * (cols + 1)] += result[ii_start_offset + (i - 1) * (cols + 1)];
			sq_result[ii_start_offset + i * (cols + 1)] += sq_result[ii_start_offset + (i - 1) * (cols + 1)];

		}
	}
}


void GpuObjDetector::GpuComputeII() {

	HANDLE_ERROR(hipMemset(dev_ii, 0, ii_mem_size));
	HANDLE_ERROR(hipMemset(dev_ii2, 0, ii_mem_size));

	dim3 block(512);
	dim3 grid_rows(ceil(img_height / 512.0));
	dim3 grid_cols(ceil(img_width / 512.0));

	kernel_ii_rows<<<grid_rows, block>>>(dev_img, dev_ii, dev_ii2, img_height, img_width);
	kernel_ii_cols<<<grid_cols, block>>>(dev_ii, dev_ii2, img_height, img_width);
}


__global__ void kernel_detect_objs(int num_stage,
								   const int *ii,
								   const int *ii2,
								   int ii_width,
								   int ii_height,
								   SubWindow *subwindows,
								   int *is_valid,
								   int num_subwindows) {
	// 244 216 123 123 6.19174

	int i_subwindow = threadIdx.x + blockIdx.x * blockDim.x;

	if (!(i_subwindow < num_subwindows)) return;

	float scale = subwindows[i_subwindow].scale;
	int x = subwindows[i_subwindow].x;
	int y = subwindows[i_subwindow].y;
	int width = subwindows[i_subwindow].w;
	int height = subwindows[i_subwindow].h;

	float inv = 1.f / (height * width);
	float mean = RectSum(ii, x, y, width, height, ii_width) * inv;
	float variance = abs(RectSum(ii2, x, y, width, height, ii_width) * inv - mean * mean);

	float std_dev = sqrtf(variance);

	Stage &stage = (Stage&)stage_buf;

	float tree_sum = 0;

	for (int j = 0; j < HAAR_MAX_TREES; j++) {
		Tree& tree = stage.trees[j];
		if (!tree.valid) break;

		float rects_sum = 0;

		for (int k = 0; k < HAAR_MAX_RECTS; k++) {
			Rectangle &rect = tree.feature.rects[k];
			if (rect.wg == 0) break;

			rects_sum = rects_sum + RectSum(ii, x + (int)(rect.x * scale),
												y + (int)(rect.y * scale),
												(int)(rect.w * scale),
												(int)(rect.h * scale), ii_width) * rect.wg;
		}

		tree_sum += ((rects_sum * inv < tree.threshold * std_dev) ? tree.left_val : tree.right_val);
	}


	is_valid[i_subwindow] = (tree_sum >= stage.threshold);

//	if (tree_sum < stage.threshold) {
//		subwindows[i_subwindow].is_object = 0;
//	}

}

bool isNonObject(const SubWindow& s) {
	return !s.is_object;
}

__global__ void kernel_compact_arrays(const SubWindow *subwindows_in,
									  SubWindow *subwindows_out,
									  int *is_valid,
									  int *indexes,
									  int num) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	if ((i < num) && is_valid[i]) {

		subwindows_out[indexes[i] - 1] = subwindows_in[i];

	}
}

inline int GetNumBlocks(int n) {
	return ceilf((float) n / MAX_THREAD);
}

void GpuObjDetector::CompactArrays(int& num_subwindows) {
	CUDPPHandle scan_plan;
	cudppPlan(lib, &scan_plan, scan_conf, num_subwindows, 1, 0);
	cudppScan(scan_plan, dev_indexes, dev_is_valid, num_subwindows);

	kernel_compact_arrays<<<GetNumBlocks(num_subwindows), MAX_THREAD>>>(dev_subwindows_in,
																		dev_subwindows_out,
																		dev_is_valid,
																		dev_indexes,
																		num_subwindows);

	hipMemcpy(&num_subwindows, (dev_indexes + num_subwindows - 1), sizeof(int), hipMemcpyDeviceToHost);

	SubWindow *tmp;
	tmp = dev_subwindows_in;
	dev_subwindows_in = dev_subwindows_out;
	dev_subwindows_out = tmp;
}

void GpuObjDetector::DetectAtSubwindows(vector<SubWindow>& subwindows) {


	int num_subwindows = subwindows.size();
	HANDLE_ERROR(hipMemcpy(dev_subwindows_in, &subwindows[0], sizeof(SubWindow) * num_subwindows, hipMemcpyHostToDevice));

	for (int i = 0; i < HAAR_MAX_STAGES; i++) {

//		int num_subwindows = subwindows.size();
		int num_blocks = ceilf((float) num_subwindows / MAX_THREAD);

//		HANDLE_ERROR(hipMemcpy(dev_subwindows_in, &subwindows[0], sizeof(SubWindow) * num_subwindows, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(stage_buf), &haar_cascade.stages[i], sizeof(Stage)));

		kernel_detect_objs<<<num_blocks, MAX_THREAD>>>(i,
											dev_ii,
											dev_ii2,
											img_width + 1,
											img_height + 1,
											dev_subwindows_in,
											dev_is_valid,
											num_subwindows);

		CompactArrays(num_subwindows);

//		HANDLE_ERROR(hipMemcpy(&subwindows[0], dev_subwindows_in, sizeof(SubWindow) * num_subwindows, hipMemcpyDeviceToHost));

//		subwindows.erase(remove_if(subwindows.begin(), subwindows.end(), isNonObject), subwindows.end());
//		DBG_WRP(cout << "Subwindows after stage " << i << " : " << subwindows.size() << endl << endl);

	}

	subwindows.resize(num_subwindows);
	HANDLE_ERROR(hipMemcpy(&subwindows[0], dev_subwindows_in, sizeof(SubWindow) * num_subwindows, hipMemcpyDeviceToHost));

}

GpuObjDetector::GpuObjDetector(int w, int h, HaarCascade& cascade) :
	img_width(w),
	img_height(h),
	haar_cascade(cascade) {

	img_mem_size = img_width * img_height * sizeof(int);
	ii_mem_size = (img_width + 1) * (img_height + 1) * sizeof(int);

	HANDLE_ERROR(hipMalloc(&dev_img, img_mem_size));
	HANDLE_ERROR(hipMalloc(&dev_ii, ii_mem_size));
	HANDLE_ERROR(hipMalloc(&dev_ii2, ii_mem_size));

	PrecalcSubwindows(img_width,
					  img_height,
					  haar_cascade.window_width,
					  haar_cascade.window_height,
					  all_subwindows);

	HANDLE_ERROR(hipMalloc(&dev_subwindows_in, sizeof(SubWindow) * all_subwindows.size()));
	HANDLE_ERROR(hipMalloc(&dev_subwindows_out, sizeof(SubWindow) * all_subwindows.size()));
	HANDLE_ERROR(hipMalloc(&dev_is_valid, sizeof(int) * all_subwindows.size()));
	HANDLE_ERROR(hipMalloc(&dev_indexes, sizeof(int) * all_subwindows.size()));

	cudppCreate(&lib);
	scan_conf.op = CUDPP_ADD;
	scan_conf.datatype = CUDPP_INT;
	scan_conf.algorithm = CUDPP_SCAN;
	scan_conf.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_INCLUSIVE;

}


void GpuObjDetector::Detect(int *g_img, std::vector<SubWindow>& objs) {

	HANDLE_ERROR(hipMemcpy(dev_img, g_img, img_mem_size, hipMemcpyHostToDevice));
	GpuComputeII();
	objs = all_subwindows;
	DetectAtSubwindows(objs);
}


GpuObjDetector::~GpuObjDetector() {
	HANDLE_ERROR(hipFree(dev_img));
	HANDLE_ERROR(hipFree(dev_ii));
	HANDLE_ERROR(hipFree(dev_ii2));
	HANDLE_ERROR(hipFree(dev_subwindows_in));
	HANDLE_ERROR(hipFree(dev_subwindows_out));
	HANDLE_ERROR(hipFree(dev_is_valid));
	HANDLE_ERROR(hipFree(dev_indexes));

}
