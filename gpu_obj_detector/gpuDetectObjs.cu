#include "hip/hip_runtime.h"
/*
 * gpuDetectObjs.cu
 *
 *  Created on: May 4, 2013
 *      Author: olehp
 */

#include "gpuDetectObjs.h"
#include "gpu_compute_ii.h"
#include "gpu_utils.h"

#include "utils.h"
#include "SubWindow.h"
#include <math.h>
#include <iostream>
#include <vector>
#include <algorithm>

using namespace std;
#define MAX_THREAD 416


__constant__ __align__(4) char c_stage[sizeof(Stage)];


__device__ inline int MatrVal(int *arr, int row, int col, int pic_width) {

	return arr[row * pic_width + col];
}

__device__ inline int RectSum(int* ii, int x, int y, int w, int h, int ii_width) {

	return MatrVal(ii, y, x, ii_width) +
		   MatrVal(ii, y + h, x + w, ii_width) -
		   MatrVal(ii, y, x + w, ii_width) -
		   MatrVal(ii, y + h, x, ii_width);
}



__global__ void kernel_ii_rows(int *matr, int *result, int *sq_result, int rows, int cols) {

	int row = threadIdx.x + blockIdx.x * blockDim.x;

	int img_start_offset = row * cols;
	int ii_start_offset = (row + 1) * (cols + 1) + 1;

	int val;
	int i;
	int cur_sum = 0, cur_sq_sum = 0;

	if (row < rows) {

		for (i = 0; i < cols; i++) {
			val = matr[img_start_offset + i];
			cur_sum += val;
			cur_sq_sum += (val * val);

			result[ii_start_offset + i] = cur_sum;
			sq_result[ii_start_offset + i] = cur_sq_sum;
		}
	}
}

__global__ void kernel_ii_cols(int *result, int *sq_result, int rows, int cols) {

	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int i;
	int ii_start_offset = (cols + 1) + col + 1;
	if (col < cols) {
		for (i = 1; i < rows; i++) {
			result[ii_start_offset + i * (cols + 1)] += result[ii_start_offset + (i - 1) * (cols + 1)];
			sq_result[ii_start_offset + i * (cols + 1)] += sq_result[ii_start_offset + (i - 1) * (cols + 1)];

		}
	}
}


void gpuComputeII(const int *matr, int **dev_result, int **dev_sq_result, int rows, int cols) {
	int *dev_matr;
	HANDLE_ERROR(hipMalloc((void **)&dev_matr, rows * cols * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void **)dev_result, (rows + 1) * (cols + 1) * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void **)dev_sq_result, (rows + 1) * (cols + 1) * sizeof(int)));
	HANDLE_ERROR(hipMemset((void *)(*dev_result), 0, (rows + 1) * (cols + 1) * sizeof(int)));
	HANDLE_ERROR(hipMemset((void *)(*dev_sq_result), 0, (rows + 1) * (cols + 1) * sizeof(int)));
	HANDLE_ERROR(hipMemcpy((void *)dev_matr, (void *)matr, rows * cols * sizeof(int), hipMemcpyHostToDevice));

	dim3 block(512);
	dim3 grid_rows(ceil(rows / 512.0));
	dim3 grid_cols(ceil(cols / 512.0));

	kernel_ii_rows<<<grid_rows, block>>>(dev_matr, *dev_result, *dev_sq_result, rows, cols);
	kernel_ii_cols<<<grid_cols, block>>>(*dev_result, *dev_sq_result, rows, cols);

	HANDLE_ERROR(hipFree(dev_matr));
}



__global__ void kernel_detect_objs(int num_stage,
								   int *ii,
								   int *ii2,
								   int ii_width,
								   int ii_height,
								   SubWindow *subwindows,
								   int num_subwindows,
								   float *num_objs) {
	// 244 216 123 123 6.19174

	int i_subwindow = threadIdx.x + blockIdx.x * blockDim.x;

	if (!(i_subwindow < num_subwindows)) return;

	float scale = subwindows[i_subwindow].scale;
	int x = subwindows[i_subwindow].x;
	int y = subwindows[i_subwindow].y;
	int width = subwindows[i_subwindow].w;
	int height = subwindows[i_subwindow].h;

	float inv = 1.0 / (height * width);
	float mean = RectSum(ii, x, y, width, height, ii_width) * inv;
	float variance = abs(RectSum(ii2, x, y, width, height, ii_width) * inv - OR_SQR(mean));

	float std_dev = sqrt(variance);

	Stage &stage = (Stage&)c_stage;

	float tree_sum = 0;

	for (int j = 0; j < HAAR_MAX_TREES; j++) {
		Tree& tree = stage.trees[j];
		if (!tree.valid) break;

		float rects_sum = 0;

		for (int k = 0; k < HAAR_MAX_RECTS; k++) {
			Rectangle &rect = tree.feature.rects[k];
			if (rect.wg == 0) break;

			rects_sum = rects_sum + RectSum(ii, x + (int)(rect.x * scale),
												y + (int)(rect.y * scale),
												(int)(rect.w * scale),
												(int)(rect.h * scale), ii_width) * rect.wg;
		}

		tree_sum += ((rects_sum * inv < tree.threshold * std_dev) ? tree.left_val : tree.right_val);
	}



	if (tree_sum < stage.threshold) {
		subwindows[i_subwindow].is_object = 0;
	}

}

bool isNonObject(const SubWindow& s) {
	return !s.is_object;
}

void detectAtSubwindows(int *dev_ii, int *dev_ii2,
						int img_width, int img_height,
						const HaarCascade& haar_cascade,
						float * dev_num_objs,
						vector<SubWindow>& subwindows) {
	float elapsed = 0;
	for (int i = 0; i < HAAR_MAX_STAGES; i++) {

		int num_subwindows = subwindows.size();
		int num_blocks = ceilf((float) num_subwindows / MAX_THREAD);

		SubWindow *dev_subwindows;
		HANDLE_ERROR(hipMalloc((void **)&dev_subwindows, sizeof(SubWindow) * num_subwindows));
		HANDLE_ERROR(hipMemcpy((void *)dev_subwindows, (void *)&subwindows[0], sizeof(SubWindow) * num_subwindows, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(c_stage), &haar_cascade.stages[i], sizeof(Stage)));

		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0);

		kernel_detect_objs<<<num_blocks, MAX_THREAD>>>(i,
											dev_ii,
											dev_ii2,
											img_width + 1,
											img_height + 1,
											dev_subwindows,
											subwindows.size(),
											dev_num_objs);

		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);

		float tmp_elapsed;
		hipEventElapsedTime(&tmp_elapsed, start, stop);

		elapsed += tmp_elapsed;
		DBG_WRP(cout << "Elapsed by stage " << tmp_elapsed << endl);

		HANDLE_ERROR(hipMemcpy((void *)&subwindows[0], (void *)dev_subwindows, sizeof(SubWindow) * num_subwindows, hipMemcpyDeviceToHost));

		subwindows.erase(remove_if(subwindows.begin(), subwindows.end(), isNonObject), subwindows.end());
		DBG_WRP(cout << "Subwindows after stage " << i << " : " << subwindows.size() << endl << endl);
		
		HANDLE_ERROR(hipFree(dev_subwindows));
	}

	DBG_WRP(cout << "Kernel elapsed: " << elapsed << endl);

}


void gpuDetectObjs(cv::Mat_<int> img,
				   const HaarCascade& haar_cascade,
				   std::vector<SubWindow>& subwindows) {
	int img_width = img.cols;
	int img_height = img.rows;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	float num_objs = 0;
	float *dev_num_objs;
	int *dev_ii;
	int *dev_ii2;
	DBG_WRP(cout << "Subwindows count: " << subwindows.size() << endl);
	DBG_WRP(cout << "Image size = " << img_width << " x " << img_height << endl);

	gpuComputeII(img.ptr<int>(), &dev_ii, &dev_ii2, img_height, img_width);

	HANDLE_ERROR(hipMalloc((void **)&dev_num_objs, sizeof(float)));

	detectAtSubwindows(dev_ii, dev_ii2, img_width, img_height, haar_cascade, dev_num_objs, subwindows);

//	HANDLE_ERROR(hipDeviceSynchronize());

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float elapsed;
	hipEventElapsedTime(&elapsed, start, stop);
	DBG_WRP(cout << "Total elapsed: " << elapsed << endl);

	hipMemcpy((void *)&num_objs, (void *)dev_num_objs, sizeof(float), hipMemcpyDeviceToHost);

//	cout << "Detected objs: " << num_objs << endl;


	HANDLE_ERROR(hipFree(dev_ii));
	HANDLE_ERROR(hipFree(dev_ii2));
	HANDLE_ERROR(hipFree(dev_num_objs));
}


















//__device__ float RectsPass(Tree &tree,
//						   int x,
//						   int y,
//						   int *ii,
//						   float scale,
//						   int img_width) {
//	float rects_sum = 0;
//	for (int i = 0; i < HAAR_MAX_RECTS; i++) {
//		Rectangle &rect = tree.feature.rects[i];
//		if (rect.wg == 0) break;
//
//		rects_sum = rects_sum +
//				RectSum(ii,
//						 x + (int)(rect.x * scale),
//						 y + (int)(rect.y * scale),
//						 (int)(rect.w * scale),
//						 (int)(rect.h * scale),
//						 img_width) * rect.wg;
//	}
//
//	return rects_sum;
//}
//
//__device__ float TreesPass(Stage &stage,
//		                   int x,
//		                   int y,
//		                   int *ii,
//		                   float scale,
//		                   float inv,
//		                   float std_dev,
//		                   int img_width) {
//
//    float tree_sum = 0;
//
//    for (int i = 0; i < HAAR_MAX_TREES; i++) {
//    	Tree& tree = stage.trees[i];
//    	if (!tree.valid) break;
//
//        float rects_sum = RectsPass(tree, x, y, ii, scale, img_width) * inv;
//
//        if (rects_sum < tree.threshold * std_dev)
//            tree_sum = tree_sum + tree.left_val;
//        else
//            tree_sum = tree_sum + tree.right_val;
//    }
//
//    return tree_sum;
//}
//
//__device__ bool StagesPass(int x,
//		                   int y,
//		                   int *ii,
//		                   float inv,
//		                   float std_dev,
//		                   float scale,
//		                   int img_width,
//		                   HaarCascade *haar_cascade) {
//
//	for (int i = 0; i < HAAR_MAX_STAGES; i++) {
//		Stage &stage = haar_cascade->stages[i];
//		if (!stage.valid) break;
//
//		float tree_sum = TreesPass(stage, x, y, ii, scale, inv, std_dev, img_width);
//		if (tree_sum < stage.threshold) {
//			return false;
//		}
//	}
//
//	return true;
//}












//bool gpuDetectObjsAt(int *ii,
//					 int *ii2,
//					 float scale,
//					 int x,
//					 int y,
//					 int width,
//					 int height,
//					 int img_width,
//					 int img_height,
//					 HaarCascade& haar_cascade) {
//
////	cout << "Starting gpuDetectObjsAt" << endl;
//
////	cout << "x = " << x << endl;
////	cout << "y = " << y << endl;
////	cout << "w = " << width << endl;
////	cout << "h = " << height << endl;
////	cout << "scale = " << scale << endl;
//
////	244 216 123 123
//	float result = 0;
//	HaarCascade *dev_haar_cascade;
//	int *dev_ii;
//	int *dev_ii2;
//	float *dev_result;
//
//
//	HANDLE_ERROR(hipMalloc((void **)&dev_result, sizeof(float)));
//	HANDLE_ERROR(hipMemcpy((void *)dev_result, (void *)&result, sizeof(float), hipMemcpyHostToDevice));
//
//	HANDLE_ERROR(hipMalloc((void **)&dev_haar_cascade, sizeof(haar_cascade)));
//	HANDLE_ERROR(hipMemcpy((void *)dev_haar_cascade, (void *)&haar_cascade, sizeof(haar_cascade), hipMemcpyHostToDevice));
//
//	HANDLE_ERROR(hipMalloc((void **)&dev_ii, sizeof(int) * img_width * img_height));
//	HANDLE_ERROR(hipMalloc((void **)&dev_ii2, sizeof(int) * img_width * img_height));
//	HANDLE_ERROR(hipMemcpy((void *)dev_ii, (void *)ii, sizeof(int) * img_width * img_height, hipMemcpyHostToDevice));
//	HANDLE_ERROR(hipMemcpy((void *)dev_ii2, (void *)ii2, sizeof(int) * img_width * img_height, hipMemcpyHostToDevice));
//
//
//	kernel_detect_objs<<<1, 1>>>(dev_ii, dev_ii2, x, y, width, height, img_width, img_height, scale, dev_haar_cascade, dev_result);
////
//	hipMemcpy((void *)&result, (void *)dev_result, sizeof(int), hipMemcpyDeviceToHost);
//
//	HANDLE_ERROR(hipFree(dev_result));
//	HANDLE_ERROR(hipFree(dev_haar_cascade));
//	HANDLE_ERROR(hipFree(dev_ii));
//	HANDLE_ERROR(hipFree(dev_ii2));
//
////	cout << "Exiting gpuDetectObjsAt" << endl;
//
//	cout << "Result: " << result << endl;
//	HANDLE_ERROR(hipDeviceSynchronize());
//
//	return result;
//}



